#include "Macro.h"
#include "Profile.h"
#include "CUAPI.h"
#include "CUDA_ConstMemory.h"

#ifdef GPU


extern int        GREP_LvUpdate;
extern int        GREPSg  [NLEVEL];
extern Profile_t *Phi_eff [NLEVEL][2];




//-------------------------------------------------------------------------------------------------------
// Function    :  CUAPI_SetConstMemory_ExtPot_GREP
// Description :  Set the constant memory variables on GPU for GREP
//
// Note        :  1. Adopt the suggested approach for CUDA version >= 5.0
//                2. Invoked by Poi_UserWorkBeforePoisson_GREP()
//                3. EXT_POT_GREP_NAUX_MAX is defined in Macro.h (default = 4000)
//
// Return      :  c_GREP_Lv_Data_New[], c_GREP_Radius[], c_GREP_NBin
//-------------------------------------------------------------------------------------------------------
void CUAPI_SetConstMemory_ExtPot_GREP()
{

   Profile_t *Phi_Lv_New = Phi_eff[GREP_LvUpdate][ GREPSg[GREP_LvUpdate] ];


   if ( Phi_Lv_New->NBin > EXT_POT_GREP_NAUX_MAX )
      Aux_Error( ERROR_INFO, "Too many bins in GREP profiles %d !!\n", Phi_Lv_New->NBin );


// transfer data only if the Profile_t object is defined
   if ( Phi_Lv_New->NBin > 0 ) {
      CUDA_CHECK_ERROR(  hipMemcpyToSymbol(HIP_SYMBOL( c_GREP_Lv_Data_New),    Phi_Lv_New->Data,   EXT_POT_GREP_NAUX_MAX*sizeof(double) )  );
      CUDA_CHECK_ERROR(  hipMemcpyToSymbol(HIP_SYMBOL( c_GREP_Lv_Radius_New),  Phi_Lv_New->Radius, EXT_POT_GREP_NAUX_MAX*sizeof(double) )  );
      CUDA_CHECK_ERROR(  hipMemcpyToSymbol(HIP_SYMBOL( c_GREP_Lv_NBin_New),   &Phi_Lv_New->NBin,                         sizeof(int   ) )  );
   }

} // FUNCTION : CUAPI_SetConstMemory_ExtPot_GREP



#endif // #ifdef GPU
