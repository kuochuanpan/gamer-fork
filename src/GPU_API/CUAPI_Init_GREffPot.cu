#include "hip/hip_runtime.h"
#include "CUAPI.h"
#include "Profile.h"

#if ( defined GPU  &&  defined GRAVITY  &&  defined GREP )


#include "CUPOT.h"

extern Profile_t *Phi_eff[2];


// declare the GPU kernel requiring GREP_Data, GREP_EdgeL, GREP_Center, and r_max2
int CUPOT_SetConstMem_GREffPot( double h_GREP_Data[], double h_GREP_Edge[], double h_GREP_Center[],
                                double h_r_max2, int h_GREP_NBin );



//-------------------------------------------------------------------------------------------------------
// Function    :  CUAPI_Init_GREffPot
// Description :  Set the auxiliary GPU constant-memory arrays for the GR effective potential

// Note        :  1. Invoked by Init_GREffPot()
//
// Parameter   :  None
//
// Return      :
//-------------------------------------------------------------------------------------------------------
void CUAPI_Init_GREffPot()
{

   int     NBin        = Phi_eff[1]->NBin;
   double *Radius      = Phi_eff[1]->Radius;
   double  r_max2      = SQR( Phi_eff[1]->MaxRadius );
   double  Edge[NBin+1];

// check
   if ( NBin > GR_POT_NAUX_MAX )
      Aux_Error( ERROR_INFO, "Too many bins in 1D Profile %d !!\n", NBin );

// compute the location of edge
   Edge[0] = 0.0;
   for ( int i=1; i<NBin; i++ )   Edge[i] = ( Phi_eff[1]->LogBin ) ? sqrt( Radius[i - 1] * Radius[i] )
                                                                   : 0.5*( Radius[i - 1] + Radius[i] );
   Edge[NBin] = ( Phi_eff[1]->LogBin ) ? SQR ( Edge[NBin - 1] ) / Edge[NBin - 2]
                                       : 2.0 * Edge[NBin - 1]   - Edge[NBin - 2];

   int Exitcode = CUPOT_SetConstMem_GREffPot( Phi_eff[1]->Data, Edge, Phi_eff[1]->Center, r_max2, NBin );
   if (  Exitcode != 0  )
      Aux_Error( ERROR_INFO, "CUPOT_SetConstMem_GREffPot failed... Exitcode %d...\n", Exitcode );

} // FUNCTION : CUAPI_Init_GREffPot



#endif // #if ( defined GPU  &&  defined GRAVITY  &&  defined GREP )
