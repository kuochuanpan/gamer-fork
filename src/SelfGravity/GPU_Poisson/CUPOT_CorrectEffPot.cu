#include "hip/hip_runtime.h"
#include "CUPOT.h"
#include <stdio.h>

#if ( defined GRAVITY  &&  defined GREP )


// external functions and GPU-related set-up
#ifdef __HIPCC__

// variables reside in constant memory
__constant__ double c_GREP_Data[GR_POT_NAUX_MAX];
__constant__ double c_GREP_Edge[GR_POT_NAUX_MAX];
__constant__ double c_GREP_Center[3];
__constant__ double c_r_max2;
__constant__ int    c_GREP_NBin;


//-------------------------------------------------------------------------------------------------------
// Function    :  CUPOT_SetConstMem_GREffPot
// Description :  Set the constant memory used by CUPOT_CorrectEffPot()
//
// Note        :  1. Adopt the suggested approach for CUDA version >= 5.0
//                2. Invoked by CUAPI_Init_GREffPot()
//
// Parameter   :  None
//
// Return      :  0/-1 : successful/failed
//---------------------------------------------------------------------------------------------------
__host__
int CUPOT_SetConstMem_GREffPot( double h_GREP_Data[], double h_GREP_Edge[], double h_GREP_Center[],
                                double h_r_max2, int h_GREP_NBin )
{

   if (  hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL( c_GREP_Data),   h_GREP_Data,   GR_POT_NAUX_MAX*sizeof(double),
                                            0, hipMemcpyHostToDevice)  )
      return -1;

   if (  hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL( c_GREP_Edge),   h_GREP_Edge,   GR_POT_NAUX_MAX*sizeof(double),
                                            0, hipMemcpyHostToDevice)  )
      return -2;

   if (  hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL( c_GREP_Center), h_GREP_Center,               3*sizeof(double),
                                            0, hipMemcpyHostToDevice)  )
      return -3;

   if (  hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL( c_r_max2),     &h_r_max2,                      sizeof(double),
                                            0, hipMemcpyHostToDevice)  )
      return -4;

   if (  hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL( c_GREP_NBin),  &h_GREP_NBin,                   sizeof(int),
                                            0, hipMemcpyHostToDevice)  )
      return -5;

   return 0;

} // FUNCTION : CUPOT_SetConstMem_GREffPot

#endif // ifdef __HIPCC__


#define LinearInterp( x, xa, xb, ya, yb )   ( ( ((x) - (xa)) * (yb) + ((xb) - (x)) * (ya) ) / ((xb) - (xa)) )




//-------------------------------------------------------------------------------------------------------
// Function    :  CUPOT_CorrectEffPot
// Description :  Do/Undo the GR potential correction
//
// Note        :  1. Support CPU only in current version
//                2. The potential correction calculated at the current step is applied to
//                   both g_Pot_Array_New and g_Pot_Array_USG in current version
//
// Parameter   :  g_Pot_Array_New   : Array storing the input potential (at the current step)
//                                    --> _New: to be distinguishable from g_Pot_Array_USG[], which is defined at the previous step
//                g_Pot_Array_USG   : Array storing the input potential for UNSPLIT_GRAVITY (at the previous step)
//                g_Corner_Array    : Array storing the physical corner coordinates of each patch
//                dh                : Cell size
//                Undo              : Add (true) or subtract (false) potential correction to the input potential
//                USG               : Flag to indicate which potential is input
//
//-------------------------------------------------------------------------------------------------------
__global__
void CUPOT_CorrectEffPot(       real   g_Pot_Array_New[][ CUBE(GRA_NXT) ],
                                real   g_Pot_Array_USG[][ CUBE(USG_NXT_G) ],
                          const double g_Corner_Array [][3],
                          const real dh, const bool Undo, const bool USG )
{

// declare index for loop
#  ifdef UNSPLIT_GRAVITY
   const int IDX    = ( USG ) ? USG_NXT_G      : GRA_NXT;
   const int IDX_GZ = ( USG ) ? USG_GHOST_SIZE : GRA_GHOST_SIZE;
#  else
   const int IDX    = GRA_NXT;
   const int IDX_GZ = GRA_GHOST_SIZE;
#  endif

   const int IDX_sqr = SQR (IDX);

   const int P = blockIdx.x;
   {
//    loop over all cells of the target patch
//    _g0: indices for the arrays without any ghost zone
      CGPU_LOOP( idx_g0, CUBE(IDX) )
      {

         const int i_g0 = idx_g0 % IDX;
         const int j_g0 = idx_g0 % IDX_sqr / IDX;
         const int k_g0 = idx_g0 / IDX_sqr;

         const double dx = g_Corner_Array[P][0] + (double)((i_g0-IDX_GZ)*dh) - c_GREP_Center[0];
         const double dy = g_Corner_Array[P][1] + (double)((j_g0-IDX_GZ)*dh) - c_GREP_Center[1];
         const double dz = g_Corner_Array[P][2] + (double)((k_g0-IDX_GZ)*dh) - c_GREP_Center[2];

         const double r2 = SQR(dx) + SQR(dy) + SQR(dz);


         if ( r2 < c_r_max2 )
         {
            const double r = SQRT( r2 );

//          use binary search algorithm to find the index of bin
            int bin;
            for ( int i=0, j=c_GREP_NBin; j - i != 1; bin = (i + j) / 2 )
            {
               int mid = (i + j) / 2;
               if ( r > c_GREP_Edge[mid] )   i = mid;
               else                          j = mid;
            }

            double phi = ( bin == c_GREP_NBin-1 ) ? c_GREP_Data[bin]
                                                  : LinearInterp( r, c_GREP_Edge[bin], c_GREP_Edge[bin+1],
                                                                     c_GREP_Data[bin], c_GREP_Data[bin+1] );

            if ( Undo )   phi = -phi;

#           ifdef UNSPLIT_GRAVITY
            if ( USG )
               g_Pot_Array_USG[P][idx_g0] += (real)phi;
            else
               g_Pot_Array_New[P][idx_g0] += (real)phi;
#           else
               g_Pot_Array_New[P][idx_g0] += (real)phi;
#           endif
         } // if ( r2 < r_max2 )

      } // CGPU_LOOP( idx_g0, CUBE(PS1) )
   } // for (int P=0; P<NPatchGroup*8; P++)

} // FUNCTION : CPU_CorrectEffPot


#endif // #if ( defined GRAVITY  &&  defined GREP )
