#include "CUAPI.h"
#include "CUFLU.h"
#include "NuclearEoS.h"

#if ( defined GPU  &&  MODEL == HYDRO  &&  EOS == EOS_NUCLEAR )


extern real *d_EoS_Table[EOS_NTABLE_MAX];




//-------------------------------------------------------------------------------------------------------
// Function    :  CUAPI_MemFree_NuclearEoS
// Description :  Free the GPU memory of the nuclear EoS
//
// Note        :  1. Invoked by EoS_End_Nuclear()
//
// Parameter   :  None
//
// Return      :  None
//-------------------------------------------------------------------------------------------------------
void CUAPI_MemFree_NuclearEoS()
{

   for (int t=0; t<NUC_TABLE_NPTR; t++)
   {
      if ( d_EoS_Table[t] != NULL ) {  CUDA_CHECK_ERROR(  hipFree( d_EoS_Table[t])  );  d_EoS_Table[t] = NULL;  }
   }

} // FUNCTION : CUAPI_MemFree_NuclearEoS



#endif // #if ( defined GPU  &&  MODEL == HYDRO  &&  EOS == EOS_NUCLEAR )
