#include "CUAPI.h"
#include "CUFLU.h"
#include "NuclearEoS.h"

#if ( defined GPU  &&  MODEL == HYDRO  &&  EOS == EOS_NUCLEAR )


extern int g_nrho;
extern int g_neps;
extern int g_nye;
extern int g_nmode;

extern real *d_EoS_Table[EOS_NTABLE_MAX];




//-------------------------------------------------------------------------------------------------------
// Function    :  CUAPI_PassNuclearEoSTable2GPU
// Description :  Allocate GPU memory and transfer data to GPU for the nuclear EoS
//
// Note        :  1. Must be invoked BEFORE calling CUAPI_SetConstMemory() to correctly set the
//                   constant-memory pointer array c_EoS_Table[]
//
// Parameter   :  None
//
// Return      :  d_EoS_Table[]
//-------------------------------------------------------------------------------------------------------
void CUAPI_PassNuclearEoSTable2GPU()
{

// set the table size
   long EoS_TableSize[NUC_TABLE_NPTR];

   EoS_TableSize[NUC_TAB_ALL      ] = sizeof(real)*g_nrho*g_neps*g_nye*NUC_TABLE_NVAR;
   EoS_TableSize[NUC_TAB_ALL_MODE ] = sizeof(real)*g_nrho*g_nmode*g_nye*3;
   EoS_TableSize[NUC_TAB_RHO      ] = sizeof(real)*g_nrho;
   EoS_TableSize[NUC_TAB_EPS      ] = sizeof(real)*g_neps;
   EoS_TableSize[NUC_TAB_YE       ] = sizeof(real)*g_nye;
   EoS_TableSize[NUC_TAB_TEMP_MODE] = sizeof(real)*g_nmode;
   EoS_TableSize[NUC_TAB_ENTR_MODE] = sizeof(real)*g_nmode;
   EoS_TableSize[NUC_TAB_PRES_MODE] = sizeof(real)*g_nmode;

   if ( MPI_Rank == 0 )
   {
      long TotalSize = 0;
      for (int t=0; t<NUC_TABLE_NPTR; t++)   TotalSize += EoS_TableSize[t];

      Aux_Message( stdout, "NOTE : total memory requirement in GPU nuclear EoS table = %ld MB\n", TotalSize/(1<<20) );
   }


// allocate GPU memory and transfer tables to GPU
// --> unlike other CPU-GPU data transfer in the code, here we do not allocate page-locked host memory (i.e., hipHostAlloc())
//     since these tables will be transferred just once
   for (int t=0; t<NUC_TABLE_NPTR; t++)
   {
      CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_EoS_Table[t], EoS_TableSize[t] )  );
      CUDA_CHECK_ERROR(  hipMemcpy( d_EoS_Table[t], h_EoS_Table[t], EoS_TableSize[t], hipMemcpyHostToDevice )  );
   }

} // FUNCTION : CUAPI_PassNuclearEoSTable2GPU



#endif // #if ( defined GPU  &&  MODEL == HYDRO  &&  EOS == EOS_NUCLEAR )
